#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define S_LEN 512
#define N 1000
#define ins -2
#define del -2
#define match -1
#define mismatch -1

#define CHECK(call)                                                     \
{                                                                       \
  const hipError_t err = call;                                         \
    if (err != hipSuccess) {                                           \
    printf("%s in %s at line %d\n", hipGetErrorString(err),            \
                                    __FILE__, __LINE__);                \
    exit(EXIT_FAILURE);                                                 \
    }                                                                   \
}
#define CHECK_KERNELCALL()                                              \
{                                                                       \
    const hipError_t err = hipGetLastError();                         \
    if (err != hipSuccess) {                                           \
        printf("%s in %s at line %d\n", hipGetErrorString(err),        \
                                        __FILE__, __LINE__);            \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
}

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

int max4(int n1, int n2, int n3, int n4)
{
    int tmp1, tmp2;
    tmp1 = n1 > n2 ? n1 : n2;
    tmp2 = n3 > n4 ? n3 : n4;
    tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
    return tmp1;
}

/*void backtrace(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
    int n;
    for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
    {
        int dir = dir_mat[i][j];
        if (dir == 1 || dir == 2)
        {
            i--;
            j--;
        }
        else if (dir == 3)
            i--;
        else if (dir == 4)
            j--;

        simple_rev_cigar[n] = dir;
    }
}*/

__global__ void kernel_gpu(char * query_hw, char * ref_hw, int * res_hw){
    unsigned int threadId = threadIdx.x;
    __shared__ int last_score[S_LEN+1];
    __shared__ int pre_score[S_LEN+1];
    //extern __shared__ int dir[];

    //First of all, I need to initialize the score matrix
    pre_score[threadId] = 0;
    last_score[threadId] = 0;
    if(threadId==1){
        pre_score[S_LEN+1] = 0;
        last_score[S_LEN+1] = 0;
    }
    __syncthreads();

    //Compute score alignment
    for(int j = 1; j<2*S_LEN; j++){
        unsigned int index = threadId;
        if(j <= S_LEN)
            index++;
        if(threadId<= j){
            unsigned int up = threadId+1;
            unsigned int left = threadId;
            unsigned int upleft = index - S_LEN;

            int tmp1, tmp2;
            int compar = (query_hw[upleft] == ref_hw[upleft]) ? match : mismatch;
            tmp1 = (pre_score[threadId] + compar) > (last_score[left] +del) ? (pre_score[threadId] + compar) : (last_score[left]+del);
            tmp2 = (last_score[up] + ins) > 0 ? (last_score[up] + ins) : 0;
            tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
            pre_score[threadId] = last_score[threadId];
            if(threadId==0)
                pre_score[S_LEN+1] = last_score[S_LEN+1];
            last_score[index] = tmp1;            
        }
        __syncthreads();
    }

    //Publish resalt on global memory
    res_hw[blockIdx.x] = last_score[0];
}

int main(int argc, char *argv[])
{
    srand(time(NULL));

    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};

    char **query = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        query[i] = (char *)malloc(S_LEN * sizeof(char));

    char **reference = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        reference[i] = (char *)malloc(S_LEN * sizeof(char));

    int **sc_mat = (int **)malloc((S_LEN + 1) * sizeof(int *));
    for (int i = 0; i < (S_LEN + 1); i++)
        sc_mat[i] = (int *)malloc((S_LEN + 1) * sizeof(int));
    /*char **dir_mat = (char **)malloc((S_LEN + 1) * sizeof(char *));
    for (int i = 0; i < (S_LEN + 1); i++)
        dir_mat[i] = (char *)malloc((S_LEN + 1) * sizeof(char));
    */
    int *res = (int *)malloc(N * sizeof(int));
    /*char **simple_rev_cigar = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        simple_rev_cigar[i] = (char *)malloc(S_LEN * 2 * sizeof(char));
    */
    int *res_hw;
    char *query_hw, *ref_hw;
    //char *cigar_hw, *cigar_sw;

    int *res_sw = (int *)malloc(N * sizeof(int));


    // Device memory allocation
    CHECK(hipMalloc(&res_hw, N*sizeof(int)));
    CHECK(hipMalloc(&query_hw, N*S_LEN*sizeof(char)));
    CHECK(hipMalloc(&ref_hw, N*S_LEN*sizeof(char)));
    //CHECK(cudaMalloc(&cigar_hw, N*2*S_LEN*sizeof(char)));


    // randomly generate sequences
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < S_LEN; j++)
        {
            query[i][j] = alphabet[rand() % 5];
            reference[i][j] = alphabet[rand() % 5];
        }
    }

    double start_cpu = get_time();

    for (int n = 0; n < N; n++)
    {
        int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
        int maxi, maxj;
        // initialize the scoring matrix and direction matrix to 0
        for (int i = 0; i < S_LEN + 1; i++)
        {
            for (int j = 0; j < S_LEN + 1; j++)
            {
                sc_mat[i][j] = 0;
                //dir_mat[i][j] = 0;
            }
        }
        // compute the alignment
        for (int i = 1; i < S_LEN + 1; i++)
        {
            for (int j = 1; j < S_LEN + 1; j++)
            {
                // compare the sequences characters
                int comparison = (query[n][i - 1] == reference[n][j - 1]) ? match : mismatch;
                // compute the cell knowing the comparison result
                int tmp = max4(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);
                /*char dir;

                if (tmp == (sc_mat[i - 1][j - 1] + comparison))
                    dir = comparison == match ? 1 : 2;
                else if (tmp == (sc_mat[i - 1][j] + del))
                    dir = 3;
                else if (tmp == (sc_mat[i][j - 1] + ins))
                    dir = 4;
                else
                    dir = 0;

                dir_mat[i][j] = dir;*/
                sc_mat[i][j] = tmp;

                if (tmp > max)
                {
                    max = tmp;
                    maxi = i;
                    maxj = j;
                }
            }
        }
        res[n] = sc_mat[maxi][maxj];
        //backtrace(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);
    }

    double end_cpu = get_time();



    // Data transmission: CPU -> GPU
    CHECK(hipMemcpy(query_hw, query, N*S_LEN*sizeof(char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(ref_hw, reference, N*S_LEN*sizeof(char), hipMemcpyHostToDevice));

    double start_gpu = get_time();
    //Kernel launch
    dim3 blocksPerGrid (N, 1, 1);
    dim3 threadsPerBlock (S_LEN, 1, 1);
    // Change for backtracking
    kernel_gpu<<<blocksPerGrid, threadsPerBlock>>>(query_hw, ref_hw, res_hw);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    double end_gpu = get_time();


    //Data transmission: GPU -> CPU
    CHECK(hipMemcpy(res_sw, res_hw, N*sizeof(char), hipMemcpyDeviceToHost));
    //CHECK(cudaMemcpy(cigar_sw, cigar_hw, N*2*S_LEN*sizeof(char), cudaMemcpyDeviceToHost));

    //Freeing memory on device
    CHECK(hipFree(query_hw));
    CHECK(hipFree(ref_hw));
    //CHECK(cudaFree(cigar_hw));
    CHECK(hipFree(res_hw));

    

    for(int i = 0; i< S_LEN; i++)
        if(res_sw[i]!=res[i]){
            printf("GPU result error!\n");
            break;
        }

    printf("SW Time CPU: %.10lf\n", end_cpu - start_cpu);
    printf("SW Time GPU: %.10lf\n", end_gpu - start_gpu);

    //Freeing host memory
    free(query);
    free(reference);
    free(sc_mat);
    free(res);
    free(res_sw);    
    
    return 0;
}